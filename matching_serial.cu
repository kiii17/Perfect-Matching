#include "hip/hip_runtime.h"
//compile using nvcc matching_serial.cu -lgsl -lgslcblas -lm


#include <stdio.h>
#include <time.h>
#include <math.h>
#include <gsl/gsl_linalg.h>
#include <gsl/gsl_sf_bessel.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_sf_bessel.h>
#include <gsl/gsl_permutation.h>
#include "timerc.h"

void randomGraph(int *graph, int dim){
	for(int i = 0; i<dim; i++){
		for(int j = i+1; j<dim; j++){
			int n = rand()%2;
			graph[i*dim+j] = n;
			graph[j*dim+i] = n;
		}
	}
}

int countEdge(int * graph, int dim){
  int count  = 0;
  for(int i = 0; i<dim*dim; i++){
    if(graph[i]>0){
      count++;
    }
  }
  return count/2;
}

void randomWeight(int * graph, int * newGraph, int dim, int edge){
  srand(time(NULL));
  for(int i = 0; i<dim; i++){
    for(int j = i; j<dim; j++){
      if(graph[i*dim+j]>0){
        newGraph[i*dim+j] = rand()%(2*edge)+1;
        newGraph[j*dim+i] = newGraph[i*dim+j];
      }else{
        newGraph[i*dim+j] = 0;
        newGraph[j*dim+i] = 0;
      }
    }
  }
}

void Tutte(int * graph, double * newGraph, int dim){
  for(int i = 0; i<dim; i++){
    for(int j = 0; j<dim; j++){
      if(graph[i*dim+j]!=0){
	if(i>j){
		newGraph[i*dim+j] = pow(2.0,(double) graph[i*dim+j]);
	}else{
		newGraph[i*dim+j] = -pow(2.0, (double) graph[i*dim+j]);
	}
      }else{
        newGraph[i*dim+j] = 0.0;
      }
    }
  }
}

void pr(int *graph, int dim){
  for(int i = 0; i<dim; i++){
    for(int j = 0; j<dim; j++){
      printf("(%d, %d) = %d\n", i, j, graph[i*dim+j]);
    }
  }
}



int main(void) {
	srand(time(0));
	//dim represents the number of vertices in the graph
	int dim = 4;
	int *graph  = (int *) malloc(dim*dim*sizeof(int));
	randomGraph(graph, dim);
	//pr prints out the adjacency matrix of the randomly generated graph
	pr(graph, dim);
	//count the number of edges in the graph
	int edge = countEdge(graph, dim);
	int *weighted = (int *)malloc(dim*dim*sizeof(int));
	//assign random weight for all edges in the graph
	randomWeight(graph, weighted, dim, edge);
	double *weighted2 = (double *)malloc(dim*dim*sizeof(double));
	//produce the Tutte matrix
	Tutte(weighted, weighted2, dim);
	//copy the Tutte matrix to a gsl_matrix
	gsl_matrix *m = gsl_matrix_alloc(dim, dim);
	for (int i = 0; i<dim; i++){
		for(int j = 0; j<dim; j++){
		gsl_matrix_set(m, i, j, weighted2[i*dim+j]);
		}
	}
	//prepare the sign and permutation matrix for LU decomposition
	int* sign = (int *)malloc(sizeof(int));
	sign[0] = (int) pow(-1, (double) dim);
	gsl_permutation *p = gsl_permutation_alloc(dim);
	gsl_linalg_LU_decomp(m, p, sign);
	//get the determinant from LU decomposition
	double d = gsl_linalg_LU_det(m, 1);
	printf("determinant is %f\n", d);
	if(d==0){
		printf("No perfect matching.\n");
		return 0;
	}
	int d1 = d;
	int b = 0;
	int k;
	//generate the weight b for the perfect matching
	while(1){
		k = d1%4;
		if(k!=0){
			break;
		} 
		d1 = d1/4;
		b++;
	}
	//printf("The weight of the perfect matching is %d\n", b);
	
	gsl_matrix *inv = gsl_matrix_alloc(dim, dim);
	//compute the inverse matrix and store it in inv
	int y = gsl_linalg_LU_invert(m, p, inv);
	//calculation to determine if each edge belong to the perfect matching
	gsl_matrix *madj = gsl_matrix_alloc (dim, dim);
	double temp;
	for (int i = 0; i<dim; i++){
		for(int j=  0; j<dim; j++){
			temp = (double)gsl_matrix_get(inv, i, j);
			gsl_matrix_set(madj, i, j, temp*abs(d1));
		}
	}
	int* result = (int *)malloc(dim*dim*sizeof(int));
	int t;
	double z;
	for (int i = 0; i<dim; i++){
		for(int j = 0; j<dim; j++){
			if(graph[i*dim+j]!=0){
				z = pow(2.0, (double) weighted[i*dim+j]);
				temp = (double)gsl_matrix_get(madj, j, i);
				temp = abs(temp)*z;
				t = (int) temp;
				if(t%2 == 1){
					result[i*dim+j] = 1;
				}
			}			
		}
	}
	printf("Result:\n");
	pr(result, dim);
	return 0;
}
