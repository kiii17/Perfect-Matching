#include "hip/hip_runtime.h"
//compile using nvcc matching_parallel.cu -lcublas

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hipblas.h"
#include "timerc.h"

void randomGraph(int *graph, int dim){
	for(int i = 0; i<dim; i++){
		for(int j = i; j<dim; j++){
			int n = rand()%2;
			graph[i*dim+j] = n;
			graph[j*dim+i] = n;
			if(i==j) graph[i*dim+j] = 0;
		}
	}
}

int countEdge(int *graph, int dim){
	int count = 0;
	for(int i = 0; i<dim*dim; i++){
		if(graph[i]>0){
			count++;
		}
	}
	return count/2;
}

void pr(int *graph, int dim){
	for(int i = 0; i<dim; i++){
		for(int j = 0; j<dim; j++){
			printf("(%d, %d) = %d\n", i, j, graph[i*dim+j]);
		}
		printf("\n");
	}
}

void prF(float *graph, int dim){
	for(int i = 0; i<dim; i++){
		for(int j = 0; j<dim; j++){
			printf("(%d, %d) = %f\n", i, j, graph[i*dim+j]);
		}
		printf("\n");
	}
}


__global__ void setup_kernel(hiprandState *state, unsigned long seed){
	int id = threadIdx.x+blockIdx.x*blockDim.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void randomWeight(int *graph, int dim, int edge, hiprandState* globalState){
	int Tutte = 0;
	if(blockIdx.x<threadIdx.x && graph[blockIdx.x*dim+threadIdx.x]!=0){	
		int id = blockIdx.x*blockDim.x+threadIdx.x;
		hiprandState localState = globalState[id];
		float RANDOM = hiprand_uniform(&localState);
		globalState[id] = localState;
		int number  = RANDOM*2*edge+1;
		graph[blockIdx.x*dim+threadIdx.x] = number;
		Tutte = (int) pow(2.0, graph[id]);
		graph[id] = Tutte;
		graph[threadIdx.x*dim+blockIdx.x] = -Tutte;	
	}
}

__global__ void adjoint(int *graph1, float *graph, int *result, int dim, float det, int weight){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	float temp = ((float)graph1[id]) *graph[id]*det;	
	int t = temp;
	if(t<0) {t = -t;}
	if(t%2==1){
		result[id] = 1;
	}else{
		result[id] = 0;
	}

}




int main(){
	srand(time(0));
	//dim represents the number of vertices in the graph
	int dim = 4;
	int *graph = (int *)malloc(dim*dim*sizeof(int));
	randomGraph(graph, dim);
	//pr prints out the adjacency matrix of the randomly generated graph
	pr(graph, dim);
	//count the number of edges in the graph
	int edge = countEdge(graph, dim);
	
	int *dev_a;
	hipMalloc((void**)&dev_a, dim*dim*sizeof(int));
	hipMemcpy(dev_a, graph, dim*dim*sizeof(int), hipMemcpyHostToDevice);

	hiprandState *devStates;
	hipMalloc(&devStates, sizeof(hiprandState));
	int seed  = rand();
	//set up random number generator
	setup_kernel<<<dim, dim>>>(devStates,seed);
	//assign random weight and produce the Tutte matrix
	randomWeight<<<dim, dim>>>(dev_a, dim, edge, devStates);	
	int *weighted = (int *)malloc(dim*dim*sizeof(int));
	hipMemcpy(weighted, dev_a, dim*dim*sizeof(int), hipMemcpyDeviceToHost);
	//copy the int matrix to a float matrix
	float *weightedF = (float *)malloc(dim*dim*sizeof(float));
	for(int i = 0; i<dim*dim; i++){
		weightedF[i] = (float) weighted[i];
	}
	float *dev_b;
	hipMalloc((void **)&dev_b, dim*dim*sizeof(float));
	hipMemcpy(dev_b, weightedF, dim*dim*sizeof(float), hipMemcpyHostToDevice);
	//allocate space and prepare for LU decompposition	
	hipblasHandle_t hdl;
	hipblasCreate(&hdl);		
	int *info;
	hipMalloc((void **)&info, sizeof(int));
	int *infoH =(int *)malloc(sizeof(int)); 
	int batch = 1;
	int *p;
	hipMalloc((void**)&p, dim*sizeof(int)); 
	float **ha = (float **)malloc(sizeof(float *));
	ha[0] = dev_b;
	float **a;
	hipMalloc((void**)&a, sizeof(float *));
	hipMemcpy(a, ha, sizeof(float *), hipMemcpyHostToDevice);
	//calculate LU decomposition
	hipblasSgetrfBatched(hdl, dim, a, dim, p, info, batch);
	hipMemcpy(infoH, info, sizeof(int), hipMemcpyDeviceToHost);	
	//copy LU decomposition to host
	hipMemcpy(weightedF, dev_b, dim*dim *sizeof(float), hipMemcpyDeviceToHost);
	//calculate determinant
	float d = 1;	
	for(int i = 0; i<dim; i++){
		d = d*weightedF[i*dim+i];
	}
	printf("det on GPU: %f\n", d);
	if(d==0){
		printf("No perfect matching.\n");
		return 0;
	}	
	int d1 =(int) d;
	int i = 1;
	int k;
	//calculate the weight i of the perfect matching
	while(1){
		k = d1%4;
		if(k!=0) break;
		d1 = d1/4;
		i++;
	}	
	//allocate space for the inverse matrix
	float **hc = (float **)malloc(sizeof(float *));
	float **c, *c1;
	hipMalloc((void **)&c, sizeof(float *));
	hipMalloc((void **)&c1, dim*dim*sizeof(float));
	hc[0] = c1;
	hipMemcpy(c, hc, sizeof(float *), hipMemcpyHostToDevice);
	//calculate the inverse matrix
	hipblasSgetriBatched(hdl, dim, a, dim, p, c, dim, info, batch);
	//copy the inverse matrix to host
	hipMemcpy(weightedF, c1, dim*dim*sizeof(float), hipMemcpyDeviceToHost);
	//allocate space for the result matrix	
	int *resultH  = (int *)malloc(dim*dim*sizeof(int));
	int *result;
	hipMalloc((void **)&result, dim*dim*sizeof(int));
	hipMemcpy(resultH, result, dim*dim*sizeof(int), hipMemcpyHostToDevice);
	//launch kernels to determine final result
	adjoint<<<dim, dim>>>(dev_a, c1, result, dim, d1, i);
	
	hipMemcpy(resultH, result, dim*dim*sizeof(int), hipMemcpyDeviceToHost);
	printf("Results:\n");
	pr(resultH, dim);
	hipFree(dev_a);
	return 0;
}
